#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 10

__global__ void suma(int* A, int* B, int* C){
	int i= threadIdx.x;
	C[i] = A[i] + B [i];
}

int main(){
	int c[N];
	{
	int a[N];
	int b[N];

	int* da;
	int* db;
	int* dc;

	for (int i = 0; i<N;i++){
		a[i]=i;
		b[i]=i*2;
		printf("%d\n",a[i]);
	}
		printf("\n");

	hipMalloc((void**)&da, sizeof(int)*N);
	hipMalloc((void**)&db, sizeof(int)*N);
	hipMalloc((void**)&dc, sizeof(int)*N);
	
	hipMemcpy(da, a, sizeof(int)*N, hipMemcpyHostToDevice);
	hipMemcpy(db, b, sizeof(int)*N, hipMemcpyHostToDevice);
	}

	suma<<<1,N>>>(da,db,dc);
	hipDeviceSynchronize();

	hipMemcpy(c, dc, sizeof(int)*N, hipMemcpyDeviceToHost);

	for (int i = 0; i<N;i++){
		printf("%d\n",c[i]);
	}
	cudaReset()

	return 0;
}

