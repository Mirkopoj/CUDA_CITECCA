#include <stdio.h>
#include <hip/hip_runtime.h>


/* Kernel */
__global__ void saludar() {

	printf("Hola Mundo desde GPU\n");
}


int main() {

	printf("Hola mundo! desde CPU \n");
	
	saludar <<<1,10>>>();
	hipDeviceReset();

	return 0;
} 




